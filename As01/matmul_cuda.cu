#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include "MatrixParser/matrixparser.h"
#include "mmio.h"

#define check(error) checkCudaCall(error, __LINE__)

const uint32_t REP = 10;
const uint32_t BLOCK_WIDTH = 16;
const uint32_t BLOCK_HEIGHT = 16;

void checkCudaCall(hipError_t result, uint32_t line)
{
    if (result != hipSuccess)
    {
        printf("cuda error \n");
        printf("Line %u: %s\n", line, hipGetErrorString(result));
        fflush(stdout);
        exit(1);
    }
}

__global__ void matrix_mul(float *A, float *B, float *C, int n, int C_width, int C_height)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= C_width || y >= C_height)
        return;

    const uint32_t xn = x * n;
    float sum = 0;

    for (int i = 0; i < n; i++)
        sum += A[xn + i] * B[y + C_width * i];

    C[x * C_width + y] = sum;
}

void gpu_memory_init(int m, int n, int p,
                     float *A, float *B, float *C,
                     float **d_A, float **d_B, float **d_C)
{
    const uint32_t A_size = sizeof(float) * n * m;
    const uint32_t B_size = sizeof(float) * p * n;
    const uint32_t C_size = sizeof(float) * p * m; 

    check(hipMalloc(d_A, A_size));
    check(hipMalloc(d_B, B_size));
    check(hipMalloc(d_C, C_size));

    check(hipMemcpy(*d_A, A, A_size, hipMemcpyHostToDevice));
    check(hipMemcpy(*d_B, B, B_size, hipMemcpyHostToDevice));
    check(hipMemcpy(*d_C, C, C_size, hipMemcpyHostToDevice));
}

void gpu_memory_free(float *d_A, float *d_B, float *d_C){
    check(hipFree(d_A));
    check(hipFree(d_B));
    check(hipFree(d_C));
}

double get_operation_count(int m, int n, int p){
    return (double)m * n * p * 9;
}

void cuda_do_compute(int m, int n, int p, float *A, float *B, float *C)
{
    float *d_A, *d_B, *d_C;
    gpu_memory_init(m, n, p, A, B, C, &d_A, &d_B, &d_C);
    const uint32_t grid_height = m % BLOCK_HEIGHT == 0 ? m / BLOCK_HEIGHT : (m / BLOCK_HEIGHT) + 1;
    const uint32_t grid_width = p % BLOCK_WIDTH == 0 ? p / BLOCK_WIDTH : (p / BLOCK_WIDTH) + 1;
    dim3 grid(grid_width, grid_height);
    dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);

    const auto start = std::chrono::system_clock::now();
    matrix_mul<<<grid, block>>>(d_A, d_B, d_C, n, p, m);
    hipDeviceSynchronize();
    const auto end = std::chrono::system_clock::now();

    const double avg_execution_time = (double)(end - start).count() / (double)10e9;
    printf("Reference code: %10.2f GFLOP/s \n", get_operation_count(m, n, p) / (avg_execution_time * 10e9));
    printf("Reference code: %10.2f seconds \n", avg_execution_time);
    hipMemcpy(C, d_C, sizeof(float) * p * m, hipMemcpyDeviceToHost);
    gpu_memory_free(d_A, d_B, d_C);
}



int main(int argc, char **argv)
{
    float *A, *B, *C;
#ifdef TIMING
    struct timeval before, after;
#endif
    int m, n, p, r, err;
    int nzA = 0, nzB = 0;
    FILE *fa, *fb, *fc;

#ifdef GENERATE
    m = M;
    n = N;
    p = P;
#else
    if (argc < 3)
    {
        fprintf(stderr, "Usage: %s [martix1] [matrix2] [resultmatrix] \n", argv[0]);
        exit(1);
    }
    else
    {
        if ((fa = fopen(argv[1], "rt")) == NULL)
            exit(1);
        if ((fb = fopen(argv[2], "rt")) == NULL)
            exit(2);
        err = read_mat(&m, &n, &p, &nzA, &nzB, fa, fb);
        if (err == -15)
        {
            printf("Matrices are incompatible! \n");
            fclose(fa);
            fclose(fb);
            exit(1);
        }
    }
#endif

    A = (float *)calloc(m * n, sizeof(float));
    if (A == NULL)
    {
        printf("Out of memory A! \n");
        exit(1);
    }
    B = (float *)calloc(n * p, sizeof(float));
    if (B == NULL)
    {
        printf("Out of memory B! \n");
        exit(1);
    }

#ifdef GENERATE
    generate_mat(m, n, p, A, B);
#else
    if (nzA > 0)
        read_sparse(fa, m, n, nzA, A);
    else
        read_dense(fa, m, n, A);
    if (nzB > 0)
        read_sparse(fb, n, p, nzB, B);
    else
        read_dense(fb, n, p, B);
    fclose(fa);
    fclose(fb);
#endif

    C = (float *)calloc(m * p, sizeof(float));
    if (C == NULL)
    {
        printf("Out of memory C1! \n");
        exit(1);
    }
    // C2 = (float *)calloc(N*P,sizeof(float));
    // if (C2==NULL) {printf("Out of memory C2! \n"); exit(1);}

    // naive implementation
#ifdef TIMING
    double flops = get_operation_count(m, n, p);
    gettimeofday(&before, NULL);
    
#endif


    for (r = 0; r < REP; r++)
        cuda_do_compute(m, n, p, A, B, C);

#ifdef TIMING
    gettimeofday(&after, NULL);
    double avg_execution_time = ((after.tv_sec + (after.tv_usec / 1000000.0)) -
                                 (before.tv_sec + (before.tv_usec / 1000000.0))) /
                                REP;
    printf("Reference code: %10.2f GFLOP/s \n", flops / (10e9 * avg_execution_time));
    printf("Reference code: %10.2f seconds \n", avg_execution_time);

#endif

#ifdef GENERATE
    if ((fc = fopen("gen_result.mtx", "wt")) == NULL)
        exit(3);
#else
    if ((fc = fopen(argv[3], "wt")) == NULL)
        exit(3);
#endif
    write_sparse(fc, m, p, C);
    fclose(fc);
    free(A);
    free(B);
    free(C);
    // free(C2);
}