#include "hip/hip_runtime.h"
/* SpMV: this file contains the I/O functions used to read and write matrices
 * in the Market Matrix format(see
 * https://math.nist.gov/MatrixMarket/formats.html#MMformat), using the
 * functions in mmio.c in its turn.
 * A vector of the appropriate size is generated and multiplied with the matrix.
 * There are also functions to generate your own matrices.
 * The reading also supports the "is_pattern" flag from the Matrix Market format. 
 */
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "mmio.h"

#define check(error) checkCudaCall(error, __LINE__)

//#define VERBOSE

#define N  512
#define M  512

#define REP 100

void checkCudaCall(hipError_t result, uint32_t line)
{
    if (result != hipSuccess)
    {
        printf("cuda error \n");
        printf("Line %u: %s\n", line, hipGetErrorString(result));
        fflush(stdout);
        exit(1);
    }
}

__global__ void csr(const int *A_rows, const int *A_cols_idx, const float *A_values, const float *B, float *C) {
    int i,j;
    int row_start, row_end;

    i = blockIdx.x;  

        float tmp = 0.0f;
        row_start = A_rows[i];
        row_end = A_rows[i + 1];
        
        for (j = row_start; j < row_end; j++) {
            tmp += A_values[j] * B[A_cols_idx[j]];
        }
        
        C[i] = tmp;
}

void gpu_memory_init(int m, int n, int nzA, int *A_rows, int *A_cols_idx, float *A_values, float *B, float *C,
    int **d_A_rows, int **d_A_cols_idx, float **d_A_values, float **d_B, float **d_C)
{
    const uint32_t A_rows_size = sizeof(int) * (m + 1);
    const uint32_t A_cols_idx_size = sizeof(int) * nzA;
    const uint32_t A_values_size = sizeof(float) * nzA;
    const uint32_t B_size = sizeof(float) * n;
    const uint32_t C_size = sizeof(float) * m; 

    check(hipMalloc(d_A_rows, A_rows_size));
    check(hipMalloc(d_A_cols_idx, A_cols_idx_size));
    check(hipMalloc(d_A_values, A_values_size));
    check(hipMalloc(d_B, B_size));
    check(hipMalloc(d_C, C_size));

    check(hipMemcpy(*d_A_rows, A_rows, A_rows_size, hipMemcpyHostToDevice));
    check(hipMemcpy(*d_A_cols_idx, A_cols_idx, A_cols_idx_size, hipMemcpyHostToDevice));
    check(hipMemcpy(*d_A_values, A_values, A_values_size, hipMemcpyHostToDevice));
    check(hipMemcpy(*d_B, B, B_size, hipMemcpyHostToDevice));
    check(hipMemcpy(*d_C, C, C_size, hipMemcpyHostToDevice));
}

void gpu_memory_free(int *d_A_rows, int *d_A_cols_idx, float *d_A_values, float *d_B, float *d_C){
    check(hipFree(d_A_rows));
    check(hipFree(d_A_cols_idx));
    check(hipFree(d_A_values));
    check(hipFree(d_B));
    check(hipFree(d_C));
}

void csr_spmv(int m, int n, int nzA, int *A_rows, int *A_cols_idx, float *A_values, float *B, float *C) {
    int r;

    int *d_A_rows, *d_A_cols_idx;
    float *d_A_values;
    float *d_B;
    float *d_C;

    const uint32_t grid_height = 1;
    const uint32_t grid_width = 1;
    const uint32_t block_height = m;
    const uint32_t block_width = 1;

    dim3 grid(grid_width, grid_height);
    dim3 block(block_width, block_height);

    gpu_memory_init(m, n, nzA, A_rows, A_cols_idx, A_values, B, C, &d_A_rows, &d_A_cols_idx, &d_A_values, &d_B, &d_C);

    const auto start = std::chrono::system_clock::now();
    for (r=0; r<REP; r++) 
    {
        csr<<<m, 1>>>(d_A_rows, d_A_cols_idx, d_A_values, d_B, d_C);
        hipDeviceSynchronize();
    }
    const auto end = std::chrono::system_clock::now();
    hipMemcpy(C, d_C, sizeof(float) * m, hipMemcpyDeviceToHost);

    using std::chrono::duration_cast;
    using std::chrono::milliseconds;
    const double avg_execution_time = duration_cast<milliseconds>(end - start).count() / 1000.0 / REP;
    printf("Seconds: %f\n", avg_execution_time);
    printf("REP: %i\n", REP);
    
    gpu_memory_free(d_A_rows, d_A_cols_idx, d_A_values, d_B, d_C);
}

/* 
 * for nz=1 - dense matrix; for nz>1 - sparse matrix with every nz element non-0; for nz=0: error
 */
void generate_mat(int m, int n, float *A, int nz) {
  int i;

  for (i=0; i<(m*n); i++) 
	A[i] = i/nz; //i/10; 
      	
}

void read_sparse(FILE *f, int m, int n, int nz, float *A, int is_pattern) {
  int i, row, col;
  float val;  
 
    /* NOTE: when reading in doubles, ANSI C requires the use of the "l"  */
    /*   specifier as in "%lg", "%lf", "%le", otherwise errors will occur */
    /*  (ANSI C X3.159-1989, Sec. 4.9.6.2, p. 136 lines 13-15)            */

    if (is_pattern) {
        for (i=0; i<nz; i++)
        {
            fscanf(f, "%d %d\n", &row, &col);
            A[(row-1)*n+col-1] = 1.0f;   /* adjust from 1-based to 0-based */
        }
    }
    else {
      for (i=0; i<nz; i++)
      {
          fscanf(f, "%d %d %f\n", &row, &col, &val);
          A[(row-1)*n+col-1] = val;   /* adjust from 1-based to 0-based */
      }
    }
}

void write_sparse(FILE *f, int m, int p, const float *C) {
   int i, nz=0; 
   MM_typecode matcode;

   for (i=0; i<m*p; i++) if (C[i] != 0.0) nz++; 

    mm_initialize_typecode(&matcode);
    mm_set_matrix(&matcode);
    mm_set_coordinate(&matcode);
    mm_set_real(&matcode);

    mm_write_banner(f, matcode); 
    mm_write_mtx_crd_size(f, m, p, nz);

    for (i=0; i<m*p; i++) {
	if (C[i] != 0.0) 
          fprintf(f, "%d %d %f\n", i/p+1, i%p+1, C[i]);
    }

}

void write_vector(FILE* f, int m, const float* C) {
  int i;

  for (i=0; i<m; i++) {
      fprintf(f, "%f\n", C[i]);
  }
}

void read_dense(FILE *f, int m, int n, float *A) {
  int row, col;

  for(row=0; row<m; row++) { 
     for (col=0; col<n; col++) {
        fscanf(f, "%f ", &A[row*n+col]); 
     }
  } 
}

void print_mat(int m, int n, float *A) {
  int row, col; 

  for(row=0; row<m; row++) {
     for (col=0; col<n; col++) {
     	printf("%10.5f", A[row*n+col]);
     }
     printf("\n"); 
  }
}


int read_mat(int *m, int *n, int *nzA, FILE* fa, int *is_pattern) {
  MM_typecode ta;
  int ret_code = 0; 

  if (mm_read_banner(fa, &ta) != 0)
    {
        printf("Could not process Matrix Market banner for A.\n");
        return -3;
    }

  if (mm_is_pattern(ta)) 
	*is_pattern = 1;
  else
	*is_pattern = 0;


  if (mm_is_complex(ta)) return -6;

  if (mm_is_matrix(ta) && mm_is_sparse(ta))
    {
        if ((ret_code = mm_read_mtx_crd_size(fa, m, n, nzA)) !=0)
           return -10;
    }
  else if (mm_is_matrix(ta) && mm_is_array(ta)) {
	*nzA = 0;
        if ((ret_code = mm_read_mtx_array_size(fa, m, n)) !=0)
           return -11;

    }
  else return -8; 

  return ret_code;
}

/*
 * Converts matrix to CSR format. 
 * returns the number of nonZero's found during conversion
 */
int convert_to_csr(int m, int n, float *A, int *sA_rows, int *sA_col_idx, float *sA_vals) {
  int i,j; 
  int checkNZ=0; 
  float tmp;
 
  sA_rows[0]=0;
  checkNZ=0;
  for (i=0; i<m; i++) {
    for (j=0; j<n; j++) {
	tmp = A[i*n+j];
	if (tmp != 0) {
 	   sA_col_idx[checkNZ]=j;
	   sA_vals[checkNZ]=tmp;
	   checkNZ++; 
	}	
    }	
    sA_rows[i+1]=checkNZ;
  }
  return checkNZ;
}

void print_mat_csr(int m, int nzA, int *sA_rows, int *sA_col_idx, float *sA_vals)
{
  int i;
  for (i=0; i<m+1; i++)
    printf("%d ", sA_rows[i]);
  printf("\n");

  for (i=0; i<nzA; i++)
    printf("%d ", sA_col_idx[i]);

  printf("\n");

  for (i=0; i<nzA; i++)
    printf("%10.5f ", sA_vals[i]);
  printf("\n");

}


int main (int argc, char** argv) {
 float *A, *B, *C;
 float *sA_vals;
 int *sA_rows, *sA_cols_idx;

 int m, n, err;
 int nzA=0, is_pattern = 1;
 FILE *fa, *fc;
  
#ifdef GENERATE 
 m=M; n=N; nzA=M*N/10; 
#else 
 if (argc < 3) {
    fprintf(stderr, "Usage: %s [matrix-market-filename] [result-vector-filename]\n", argv[0]);
    exit(1);
 }
 else {
    if ((fa = fopen(argv[1], "rt")) == NULL) exit(1);
    err = read_mat(&m, &n, &nzA, fa, &is_pattern);    
    if (err == -15) {
	printf("Matrices are incompatible! \n");
	fclose(fa); 
	exit(1);
    }
 }
#endif

 A = (float *)calloc(m*n,sizeof(float));
 if (A==NULL) {printf("Out of memory A! \n"); exit(1);}

#ifdef GENERATE
   generate_mat(m,n,A,nzA);
#else 
   if (nzA>0) {
	read_sparse(fa, m,n,nzA, A, is_pattern);
    }	
   else 
	read_dense(fa, m,n, A);
   fclose(fa);

        sA_rows = (int *)calloc(m+1,sizeof(int));
        sA_cols_idx = (int *)calloc(nzA,sizeof(int));
        sA_vals = (float *)calloc(nzA,sizeof(float));

        convert_to_csr(m,n, A, sA_rows, sA_cols_idx, sA_vals);
#ifdef VERBOSE
	print_mat(m,n,A);	
 	print_mat_csr(m, nzA, sA_rows, sA_cols_idx, sA_vals);
#endif

#endif

 B = (float *)calloc(n,sizeof(float));
 if (B==NULL) {printf("Out of memory B! \n"); exit(1);}

   generate_mat(n,1,B,1);
#ifdef VERBOSE
   print_mat(n,1,B);
#endif 

 C = (float *)calloc(m,sizeof(float));
 if (C==NULL) {printf("Out of memory C! \n"); exit(1);}

 /* Call the SpMV kernel. */
  csr_spmv(m, n, nzA, sA_rows, sA_cols_idx, sA_vals, B, C);

 if ((fc = fopen(argv[2], "wt")) == NULL) exit(3);    
// write_sparse(fc,n,m,C);
 write_vector(fc,m,C);
 fclose(fc);  

 free(A);
 free(B); 
 free(C);

}

