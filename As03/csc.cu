#include "hip/hip_runtime.h"
/* SpMV: this file contains the I/O functions used to read and write matrices
 * in the Market Matrix format(see
 * https://math.nist.gov/MatrixMarket/formats.html#MMformat), using the
 * functions in mmio.c in its turn.
 * A vector of the appropriate size is generated and multiplied with the matrix.
 * There are also functions to generate your own matrices.
 * The reading also supports the "is_pattern" flag from the Matrix Market format. 
 */
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "mmio.h"

#define check(error) checkCudaCall(error, __LINE__)

//#define VERBOSE

#define N  512
#define M  512

#define REP 100

void checkCudaCall(hipError_t result, uint32_t line)
{
    if (result != hipSuccess)
    {
        printf("cuda error \n");
        printf("Line %u: %s\n", line, hipGetErrorString(result));
        fflush(stdout);
        exit(1);
    }
}

__global__ void csc(const int *A_cols, const int *A_rows_idx, const float *A_values, const float *B, float *C) {
    int i,j;
    int col_start, col_end; 

    i = blockIdx.x;  

        col_start = A_cols[i];
        col_end = A_cols[i + 1];
        
        for (j = col_start; j < col_end; j++) {
            C[A_rows_idx[j]] += A_values[j] * B[i];
        }
}

void gpu_memory_init(int m, int n, int nzA, int *A_cols, int *A_rows_idx, float *A_values, float *B, float *C,
    int **d_A_cols, int **d_A_rows_idx, float **d_A_values, float **d_B, float **d_C)
{
    const uint32_t A_cols_size = sizeof(int) * (n + 1);
    const uint32_t A_rows_idx_size = sizeof(int) * nzA;
    const uint32_t A_values_size = sizeof(float) * nzA;
    const uint32_t B_size = sizeof(float) * n;
    const uint32_t C_size = sizeof(float) * m; 

    check(hipMalloc(d_A_cols, A_cols_size));
    check(hipMalloc(d_A_rows_idx, A_rows_idx_size));
    check(hipMalloc(d_A_values, A_values_size));
    check(hipMalloc(d_B, B_size));
    check(hipMalloc(d_C, C_size));

    check(hipMemcpy(*d_A_cols, A_cols, A_cols_size, hipMemcpyHostToDevice));
    check(hipMemcpy(*d_A_rows_idx, A_rows_idx, A_rows_idx_size, hipMemcpyHostToDevice));
    check(hipMemcpy(*d_A_values, A_values, A_values_size, hipMemcpyHostToDevice));
    check(hipMemcpy(*d_B, B, B_size, hipMemcpyHostToDevice));
    check(hipMemcpy(*d_C, C, C_size, hipMemcpyHostToDevice));
}

void gpu_memory_free(int *d_A_cols, int *d_A_rows_idx, float *d_A_values, float *d_B, float *d_C){
    check(hipFree(d_A_cols));
    check(hipFree(d_A_rows_idx));
    check(hipFree(d_A_values));
    check(hipFree(d_B));
    check(hipFree(d_C));
}

void csc_spmv(int m, int n, int nzA, int *A_cols, int *A_rows_idx, float *A_values, float *B, float *C) {
    int r;

    int *d_A_cols, *d_A_rows_idx;
    float *d_A_values;
    float *d_B;
    float *d_C;

    gpu_memory_init(m, n, nzA, A_cols, A_rows_idx, A_values, B, C, &d_A_cols, &d_A_rows_idx, &d_A_values, &d_B, &d_C);

    const auto start = std::chrono::system_clock::now();
    for (r=0; r<REP; r++) 
    {
        csc<<<n, 1>>>(d_A_cols, d_A_rows_idx, d_A_values, d_B, d_C);
        hipDeviceSynchronize();
    }
    const auto end = std::chrono::system_clock::now();
    hipMemcpy(C, d_C, sizeof(float) * m, hipMemcpyDeviceToHost);

    using std::chrono::duration_cast;
    using std::chrono::milliseconds;
    const double avg_execution_time = duration_cast<milliseconds>(end - start).count() / 1000.0 / REP;
    printf("Seconds: %f\n", avg_execution_time);
    printf("REP: %i\n", REP);
    
    gpu_memory_free(d_A_cols, d_A_rows_idx, d_A_values, d_B, d_C);
}

/* 
 * for nz=1 - dense matrix; for nz>1 - sparse matrix with every nz element non-0; for nz=0: error
 */
void generate_mat(int m, int n, float *A, int nz) {
  int i;

  for (i=0; i<(m*n); i++) 
	A[i] = i/nz; //i/10; 
      	
}

void read_sparse(FILE *f, int m, int n, int nz, float *A, int is_pattern) {
  int i, row, col;
  float val;  
 
    /* NOTE: when reading in doubles, ANSI C requires the use of the "l"  */
    /*   specifier as in "%lg", "%lf", "%le", otherwise errors will occur */
    /*  (ANSI C X3.159-1989, Sec. 4.9.6.2, p. 136 lines 13-15)            */

    if (is_pattern) {
        for (i=0; i<nz; i++)
        {
            fscanf(f, "%d %d\n", &row, &col);
            A[(row-1)*n+col-1] = 1.0f;   /* adjust from 1-based to 0-based */
        }
    }
    else {
      for (i=0; i<nz; i++)
      {
          fscanf(f, "%d %d %f\n", &row, &col, &val);
          A[(row-1)*n+col-1] = val;   /* adjust from 1-based to 0-based */
      }
    }
}

void write_sparse(FILE *f, int m, int p, const float *C) {
   int i, nz=0; 
   MM_typecode matcode;

   for (i=0; i<m*p; i++) if (C[i] != 0.0) nz++; 

    mm_initialize_typecode(&matcode);
    mm_set_matrix(&matcode);
    mm_set_coordinate(&matcode);
    mm_set_real(&matcode);

    mm_write_banner(f, matcode); 
    mm_write_mtx_crd_size(f, m, p, nz);

    for (i=0; i<m*p; i++) {
	if (C[i] != 0.0) 
          fprintf(f, "%d %d %f\n", i/p+1, i%p+1, C[i]);
    }

}

void write_vector(FILE* f, int m, const float* C) {
  int i;

  for (i=0; i<m; i++) {
      fprintf(f, "%f\n", C[i]);
  }
}

void read_dense(FILE *f, int m, int n, float *A) {
  int row, col;

  for(row=0; row<m; row++) { 
     for (col=0; col<n; col++) {
        fscanf(f, "%f ", &A[row*n+col]); 
     }
  } 
}

void print_mat(int m, int n, float *A) {
  int row, col; 

  for(row=0; row<m; row++) {
     for (col=0; col<n; col++) {
     	printf("%10.5f", A[row*n+col]);
     }
     printf("\n"); 
  }
}


int read_mat(int *m, int *n, int *nzA, FILE* fa, int *is_pattern) {
  MM_typecode ta;
  int ret_code = 0; 

  if (mm_read_banner(fa, &ta) != 0)
    {
        printf("Could not process Matrix Market banner for A.\n");
        return -3;
    }

  if (mm_is_pattern(ta)) 
	*is_pattern = 1;
  else
	*is_pattern = 0;


  if (mm_is_complex(ta)) return -6;

  if (mm_is_matrix(ta) && mm_is_sparse(ta))
    {
        if ((ret_code = mm_read_mtx_crd_size(fa, m, n, nzA)) !=0)
           return -10;
    }
  else if (mm_is_matrix(ta) && mm_is_array(ta)) {
	*nzA = 0;
        if ((ret_code = mm_read_mtx_array_size(fa, m, n)) !=0)
           return -11;

    }
  else return -8; 

  return ret_code;
}

/*
 *  * Converts matrix to CSC format. 
 *   * returns the number of nonZero's found during conversion
 *    */
int convert_to_csc(int m, int n, float *A, int *sA_cols, int *sA_row_idx, float *sA_vals) {
  int i,j;
  int checkNZ=0;
  float tmp;

  sA_cols[0]=0;
  checkNZ=0;
  for (i=0; i<n; i++) {
    for (j=0; j<m; j++) {
        tmp = A[j*n+i];
        if (tmp != 0) {
           sA_row_idx[checkNZ]=j;
           sA_vals[checkNZ]=tmp;
           checkNZ++;
        }
    }
    sA_cols[i+1]=checkNZ;
  }
  return checkNZ;
}

void print_mat_csc(int n, int nzA, int *sA_cols, int *sA_row_idx, float *sA_vals)
{
  int i;
  for (i=0; i<n+1; i++)
    printf("%d ", sA_cols[i]);
  printf("\n");

  for (i=0; i<nzA; i++)
    printf("%d ", sA_row_idx[i]);

  printf("\n");

  for (i=0; i<nzA; i++)
    printf("%10.5f ", sA_vals[i]);
  printf("\n");

}


int main (int argc, char** argv) {
 float *A, *B, *C;
 float *sA_vals;
 int *sA_cols, *sA_rows_idx;

 int m, n, err;
 int nzA=0, is_pattern = 1;
 FILE *fa, *fc;
  
#ifdef GENERATE 
 m=M; n=N; nzA=M*N/10; 
#else 
 if (argc < 3) {
    fprintf(stderr, "Usage: %s [matrix-market-filename] [result-vector-filename]\n", argv[0]);
    exit(1);
 }
 else {
    if ((fa = fopen(argv[1], "rt")) == NULL) exit(1);
    err = read_mat(&m, &n, &nzA, fa, &is_pattern);    
    if (err == -15) {
	printf("Matrices are incompatible! \n");
	fclose(fa); 
	exit(1);
    }
 }
#endif

 A = (float *)calloc(m*n,sizeof(float));
 if (A==NULL) {printf("Out of memory A! \n"); exit(1);}

#ifdef GENERATE
   generate_mat(m,n,A,nzA);
#else 
   if (nzA>0) {
	read_sparse(fa, m,n,nzA, A, is_pattern);
    }	
   else 
	read_dense(fa, m,n, A);
   fclose(fa);

        sA_cols = (int *)calloc(n+1,sizeof(int));
        sA_rows_idx = (int *)calloc(nzA,sizeof(int));
        sA_vals = (float *)calloc(nzA,sizeof(float));

        convert_to_csc(m,n, A, sA_cols, sA_rows_idx, sA_vals);
#ifdef VERBOSE
        print_mat(m,n,A);
        print_mat_csc(m, nzA, sA_cols, sA_rows_idx, sA_vals);
#endif

#endif

 B = (float *)calloc(n,sizeof(float));
 if (B==NULL) {printf("Out of memory B! \n"); exit(1);}

   generate_mat(n,1,B,1);
#ifdef VERBOSE
   print_mat(n,1,B);
#endif 

 C = (float *)calloc(m,sizeof(float));
 if (C==NULL) {printf("Out of memory C! \n"); exit(1);}

 /* Call the SpMV kernel. */
  csc_spmv(m, n, nzA, sA_cols, sA_rows_idx, sA_vals, B, C);

 if ((fc = fopen(argv[2], "wt")) == NULL) exit(3);    
// write_sparse(fc,n,m,C);
 write_vector(fc,m,C);
 fclose(fc);  

 free(A);
 free(B); 
 free(C);

}

